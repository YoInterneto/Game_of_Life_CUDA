#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include "./common/book.h"

#include <time.h>
#include <stdio.h>
#include <iostream>
#include <stdlib.h>

#define TILE_WIDTH 2

void imprimirMatriz(char* matriz, int dimension, int columna);

void rellenarMatriz(char* matriz, int dimension);

int contarVivas(char* matriz, int dimension);

hipError_t lanzarKernel(char* matriz, char* matrizResultado, int fila, int columna);

__global__ void movimientoCelularBloque(char* matriz, char* matrizResultado, int fila, int columna);


int main(int arg, char* argv[])
{

    //Comprueba que haya solo el numero de argumento permitidos
    if (arg != 4) {
        printf("\nERROR: El numero de argumentos es erroneo (.exe <-a/-m> <fila> <columna>)\n");
    }
    else {

        //Conversion de argumentos a int
        char* filaPuntero = argv[2];
        int fila = atoi(filaPuntero);
        char* columnaPuntero = argv[3];
        int columna = atoi(columnaPuntero);

        //Inicializamos hipDeviceProp_t para coger las propiedades de la tarjeta
        hipDeviceProp_t propiedades;
        HANDLE_ERROR(hipGetDeviceProperties(&propiedades, 0));

        //Dimension de la matriz
        int dimension = columna * fila;

        //Matrices
        char* matriz = NULL;
        char* matrizResultado = NULL;

        matriz = (char*) malloc(sizeof(char) * dimension);
        matrizResultado = (char*) malloc(sizeof(char) * dimension);

        //Booleano para saber si el usuario quiere manual o automatico, por defecto automatico
        bool manual = false;

        //Comprueba que los numeros de columna y fila son correctos
        if (columna <= 0 | fila <= 0) {
            printf("\nERROR: La fila/columna tiene que ser un entero positivo.\n");
        }
        //Comprueba que se haya introducido el parametro de ejecucion correcto 
        else if ((strcmp("-m", argv[1]) & strcmp("-a", argv[1])) != 0) {
            printf("\nERROR: Argumentos validos solo -m[manual] o -a[automatico]\n");
        }
        //Una vez comprobado todo empezamos con la ejecucion
        else {

            printf("\n[Matriz(%dx%d) Dimension(%d)] [modo: %s] \n", fila, columna, dimension, argv[1]);

            if (strcmp("-m", argv[1]) == 0) {
                manual = true;
            }

            //Rellenamos el tablero con celulas muertas y vivas
            rellenarMatriz(matriz, dimension);

            printf("\n***TABLERO INICIAL***\n");
            imprimirMatriz(matriz, dimension, columna);

            int generaciones = 1;
            int vivas = 0;

            while (vivas != dimension) {
                 
                system("CLS");

                if (generaciones == 1) {
                    lanzarKernel(matriz, matrizResultado, fila, columna);
                }
                else {
                    lanzarKernel(matrizResultado, matrizResultado, fila, columna);
                }

                vivas = contarVivas(matrizResultado, dimension);

                printf("\nGeneracion: %d\n", generaciones);
                printf("Celulas vivas: %d\n", vivas);
                imprimirMatriz(matrizResultado, dimension, columna);

                if (manual) {
                    system("pause");
                }
                else {
                    Sleep(1000);
                }

                generaciones++;
            }
        }

        //Liberamos los arrays
        free(matriz);
        free(matrizResultado);

    }
}

hipError_t lanzarKernel(char* matriz, char* matrizResultado, int fila, int columna) {

    //Punteros a las matrices que se meten por el kernel
    char* matriz_d;
    char* matrizResultado_d;

    int dimension = fila * columna;

    hipError_t cudaStatus;

    //Dimension del bloque y grid
    dim3 dimGrid(fila/TILE_WIDTH, columna/TILE_WIDTH);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);

    //Seleccionamos el device
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice fallo: Tienes una GPU instalada?");
        goto Error;
    }

    //Reservamos las memorias
    cudaStatus = hipMalloc((void**)&matriz_d, dimension * sizeof(char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "ERROR: hipMalloc matriz_d fallo.");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&matrizResultado_d, dimension * sizeof(char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "ERROR: hipMalloc matrizResultado_d fallo.");
        goto Error;
    }

    //Copiamos las matrices que entran por parametro
    cudaStatus = hipMemcpy(matriz_d, matriz, dimension * sizeof(char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "ERROR: hipMemcpy matriz a matriz_d fallo.");
        goto Error;
    }

    cudaStatus = hipMemcpy(matrizResultado_d, matrizResultado, dimension * sizeof(char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "ERROR: hipMemcpy matrizResultado a matrizResultado_d fallo.");
        goto Error;
    }


    //Lanzamos el kernel
    movimientoCelularBloque <<< dimGrid, dimBlock >>> (matriz_d, matrizResultado_d, fila, columna);


    //Miramos los errores al lanzar el kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "ERROR: lanzamiento de kernel fallo: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    //Miramos errores despues de lanzar el kernel
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "ERROR: el kernel fallo con codigo %d\n", cudaStatus);
        goto Error;
    }

    //Copiamos el resultado en nuestra matriz
    cudaStatus = hipMemcpy(matrizResultado, matrizResultado_d, dimension * sizeof(char), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "ERROR: hipMemcpy matrizResultado_d a matrizResultado fallo.");
        goto Error;
    }


Error:
    hipFree(matriz_d);
    hipFree(matrizResultado_d);

    return cudaStatus;
}

__global__ void movimientoCelularBloque(char* matriz, char* matrizResultado, int fila, int columna) {

    int filaPos = blockIdx.x * blockDim.x + threadIdx.x;
    int columnaPos = blockIdx.y * blockDim.y + threadIdx.y;

    int posicion = filaPos * columna + columnaPos;

    //printf("BLOCK[%dx%d] DIM[%dx%d] POSICION[%dx%d] -> %d\n", blockIdx.x, blockIdx.y, blockDim.x, blockDim.y, threadIdx.x, threadIdx.y,posicion);

    int contador = 0;

    //Primera fila 0x
    if (filaPos == 0) {
        //Posicion esquina ariba izquierda 0x0
        if (columnaPos == 0) {

            if ((matriz[posicion + 1]) == 'X') { contador++; }
            if ((matriz[posicion + columna]) == 'X') { contador++; }
            if ((matriz[posicion + (columna + 1)]) == 'X') { contador++; }

            //VIVA
            if (matriz[posicion] == 'X') {

                if (contador >= 2) { matrizResultado[posicion] = 'X'; }
                else { matrizResultado[posicion] = 'O'; }
            }
            //MUERTA
            else {

                if (contador >= 3) { matrizResultado[posicion] = 'X'; }
                else { matrizResultado[posicion] = 'O'; }
            }
        }
        //Posicion esquina superior derecha
        else if (columnaPos == (columna - 1)) {

            if ((matriz[posicion - 1]) == 'X') { contador++; }
            if ((matriz[posicion + columna]) == 'X') { contador++; }
            if ((matriz[posicion + (columna - 1)]) == 'X') { contador++; }

            //VIVA
            if (matriz[posicion] == 'X') {

                if (contador >= 2) { matrizResultado[posicion] = 'X'; }
                else { matrizResultado[posicion] = 'O'; }
            }
            //MUERTA
            else {

                if (contador >= 3) { matrizResultado[posicion] = 'X'; }
                else { matrizResultado[posicion] = 'O'; }
            }
        }
        //Posicion en la primera fila sin contar esquinas
        else {

            if ((matriz[posicion - 1]) == 'X') { contador++; }
            if ((matriz[posicion + 1]) == 'X') { contador++; }
            if ((matriz[posicion + columna]) == 'X') { contador++; }
            if ((matriz[posicion + (columna - 1)]) == 'X') { contador++; }
            if ((matriz[posicion + (columna + 1)]) == 'X') { contador++; }

            //VIVA
            if (matriz[posicion] == 'X') {

                if (contador >= 2) { matrizResultado[posicion] = 'X'; }
                else { matrizResultado[posicion] = 'O'; }
            }
            //MUERTA
            else {

                if (contador >= 3) { matrizResultado[posicion] = 'X'; }
                else { matrizResultado[posicion] = 'O'; }
            }
        }
    }
    //****************************
    //Ulima fila finalXx
    else if (filaPos == (fila - 1)) {
        //Posicion esquina abajo izquierda
        if (columnaPos == 0) {

            if ((matriz[posicion + 1]) == 'X') { contador++; }
            if ((matriz[posicion - columna]) == 'X') { contador++; }
            if ((matriz[posicion - (columna - 1)]) == 'X') { contador++; }

            //VIVA
            if (matriz[posicion] == 'X') {

                if (contador >= 2) { matrizResultado[posicion] = 'X'; }
                else { matrizResultado[posicion] = 'O'; }
            }
            //MUERTA
            else {

                if (contador >= 3) { matrizResultado[posicion] = 'X'; }
                else { matrizResultado[posicion] = 'O'; }
            }
        }
        //Posicion esquina abajo derecha
        else if (columnaPos == (columna - 1)) {

            if ((matriz[posicion - 1]) == 'X') { contador++; }
            if ((matriz[posicion - columna]) == 'X') { contador++; }
            if ((matriz[posicion - (columna + 1)]) == 'X') { contador++; }

            //VIVA
            if (matriz[posicion] == 'X') {

                if (contador >= 2) { matrizResultado[posicion] = 'X'; }
                else { matrizResultado[posicion] = 'O'; }
            }
            //MUERTA
            else {

                if (contador >= 3) { matrizResultado[posicion] = 'X'; }
                else { matrizResultado[posicion] = 'O'; }
            }
        }
        //Posiciones ultima fila entre esquinas
        else {

            if ((matriz[posicion - 1]) == 'X') { contador++; }
            if ((matriz[posicion + 1]) == 'X') { contador++; }
            if ((matriz[posicion - columna]) == 'X') { contador++; }
            if ((matriz[posicion - (columna + 1)]) == 'X') { contador++; }
            if ((matriz[posicion - (columna - 1)]) == 'X') { contador++; }

            //VIVA
            if (matriz[posicion] == 'X') {

                if (contador >= 2) { matrizResultado[posicion] = 'X'; }
                else { matrizResultado[posicion] = 'O'; }
            }
            //MUERTA
            else {

                if (contador >= 3) { matrizResultado[posicion] = 'X'; }
                else { matrizResultado[posicion] = 'O'; }
            }
        }
    }
    //****************************
    //Primera columna entre las dos esquinas izquierdas
    else if (columnaPos == 0) {

        if ((matriz[posicion + 1]) == 'X') { contador++; }
        if ((matriz[posicion - columna]) == 'X') { contador++; }
        if ((matriz[posicion + columna]) == 'X') { contador++; }
        if ((matriz[posicion + (columna + 1)]) == 'X') { contador++; }
        if ((matriz[posicion - (columna - 1)]) == 'X') { contador++; }

        //VIVA
        if (matriz[posicion] == 'X') {

            if (contador >= 2) { matrizResultado[posicion] = 'X'; }
            else { matrizResultado[posicion] = 'O'; }
        }
        //MUERTA
        else {

            if (contador >= 3) { matrizResultado[posicion] = 'X'; }
            else { matrizResultado[posicion] = 'O'; }
        }
    }
    //****************************
    //Ultima colunmna xfinalY
    else if (columnaPos == columna - 1) {

        if ((matriz[posicion - 1]) == 'X') { contador++; }
        if ((matriz[posicion + columna]) == 'X') { contador++; }
        if ((matriz[posicion - columna]) == 'X') { contador++; }
        if ((matriz[posicion - (columna + 1)]) == 'X') { contador++; }
        if ((matriz[posicion + (columna - 1)]) == 'X') { contador++; }

        //VIVA
        if (matriz[posicion] == 'X') {

            if (contador >= 2) { matrizResultado[posicion] = 'X'; }
            else { matrizResultado[posicion] = 'O'; }
        }
        //MUERTA
        else {

            if (contador >= 3) { matrizResultado[posicion] = 'X'; }
            else { matrizResultado[posicion] = 'O'; }
        }
    }
    //****************************
    //Posiciones fuera de los margenes
    else {

        if ((matriz[posicion + 1]) == 'X') { contador++; }
        if ((matriz[posicion - 1]) == 'X') { contador++; }
        if ((matriz[posicion + columna]) == 'X') { contador++; }
        if ((matriz[posicion - columna]) == 'X') { contador++; }
        if ((matriz[posicion - (columna + 1)]) == 'X') { contador++; }
        if ((matriz[posicion - (columna - 1)]) == 'X') { contador++; }
        if ((matriz[posicion + (columna + 1)]) == 'X') { contador++; }
        if ((matriz[posicion + (columna - 1)]) == 'X') { contador++; }

        //VIVA
        if (matriz[posicion] == 'X') {

            if (contador >= 2) { matrizResultado[posicion] = 'X'; }
            else { matrizResultado[posicion] = 'O'; }
        }
        //MUERTA
        else {

            if (contador >= 3) { matrizResultado[posicion] = 'X'; }
            else { matrizResultado[posicion] = 'O'; }
        }
    }
}

void imprimirMatriz(char* matriz, int dimension, int columna) {

    for (int i = 0; i < dimension; i++) {

        if (matriz[i] == 'X') {
            printf(" 0 ");
        }
        else {
            printf(" . ");
        }

        if ((i + 1) % columna == 0) {
            printf("\n");
        }
    }
}

int contarVivas(char* matriz, int dimension) {

    int contador = 0;

    for (int i = 0; i < dimension; i++) {
        if (matriz[i] == 'X') {
            contador++;
        }
    }

    return contador;
}

void rellenarMatriz(char* matriz, int dimension) {

    srand(time(0));

    for (int i = 0; i < dimension; i++) {

        char* celula = matriz + i;

        int random = rand() % dimension + 1;

        if (random % 3 == 0 && random % 2 == 0) {

            *celula = 'X';
        }
        else {
            *celula = 'O';
        }

    }
}